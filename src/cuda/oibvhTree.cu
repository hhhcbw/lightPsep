#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>

#include "cuda/oibvhTree.cuh"
#include "cuda/oibvh.cuh"
#include "cuda/utils.cuh"

oibvhTree::oibvhTree(const std::shared_ptr<Mesh> mesh) : m_mesh(mesh)
{
    setup();
}

void oibvhTree::setup()
{
    std::cout << "--set up--" << std::endl;
    for (int i = 0; i < m_mesh->m_facesCount; i++)
    {
        m_faces.push_back(
            glm::uvec3(m_mesh->m_indices[i * 3], m_mesh->m_indices[i * 3 + 1], m_mesh->m_indices[i * 3 + 2]));
    }
    for (auto vertex : m_mesh->m_vertices)
    {
        m_positions.push_back(vertex.m_position);
    }
    std::cout << "faces count: " << m_faces.size() << std::endl;
    std::cout << "vertices count: " << m_positions.size() << std::endl;
}

void oibvhTree::build()
{
    std::cout << "--build oibvh tree--" << std::endl;
    int dev;
    float elapsed_ms;
    hipGetDevice(&dev);
    std::cout << "device id: " << dev << std::endl;
    const unsigned int primitive_count = m_faces.size();
    const unsigned int vertex_count = m_positions.size();
    // std::cout << oibvh_get_size(2147483647) << std::endl;
    const unsigned int oibvh_size = oibvh_get_size(primitive_count);
    const unsigned int oibvh_internal_node_count = oibvh_size - primitive_count;
    glm::vec3* d_positions;
    glm::uvec3* d_faces;
    aabb_box_t* d_aabbs;
    unsigned int* d_mortons;
    deviceMalloc(&d_positions, vertex_count);
    deviceMalloc(&d_faces, primitive_count);
    deviceMalloc(&d_aabbs, oibvh_size);
    deviceMalloc(&d_mortons, primitive_count);
    deviceMemcpy(d_positions, &m_positions[0], vertex_count);
    deviceMemcpy(d_faces, &m_faces[0], primitive_count);

    elapsed_ms = kernelLaunch([&]() {
        dim3 blockSize = dim3(256);
        int bx = (primitive_count + blockSize.x - 1) / blockSize.x;
        dim3 gridSize = dim3(bx);
        calculate_aabb_and_morton_kernel<<<gridSize, blockSize>>>(
            d_faces, d_positions, primitive_count, m_mesh->m_aabb, d_aabbs + oibvh_internal_node_count, d_mortons);
    });
    std::cout << "AABBs and mortons calculation took: " << elapsed_ms << "ms" << std::endl;

#if 0
     // check result
     aabb_box_t* temp_aabbs;
     hostMalloc(&temp_aabbs, primitive_count);
     hostMemcpy(temp_aabbs, d_aabbs + oibvh_internal_node_count, primitive_count);
     aabb_box_t aabb;
     std::cout << sizeof(aabb_box_t) << std::endl;
     aabb.minimum = glm::vec3(1e10);
     aabb.maximum = glm::vec3(-1e10);
     for (int i = 0; i < primitive_count; i++)
    {
        aabb.maximum = glm::max(aabb.maximum, temp_aabbs[i].maximum);
        aabb.minimum = glm::min(aabb.minimum, temp_aabbs[i].minimum);
    }
     aabb == m_mesh->m_aabb ? std::cout << "aabb is correct" << std::endl : std::cout << "aabb is wrong" << std::endl;
     delete[] temp_aabbs;
#endif

    thrust::device_ptr<unsigned int> d_mortons_ptr(d_mortons);
    thrust::device_ptr<glm::uvec3> d_faces_ptr(d_faces);
    thrust::device_ptr<aabb_box_t> d_aabbs_ptr(d_aabbs + oibvh_internal_node_count);
    elapsed_ms = kernelLaunch([&]() {
        thrust::stable_sort_by_key(d_mortons_ptr, d_mortons_ptr + primitive_count, d_faces_ptr);
        thrust::stable_sort_by_key(d_mortons_ptr, d_mortons_ptr + primitive_count, d_aabbs_ptr);
    });
    std::cout << "Sorting took: " << elapsed_ms << "ms" << std::endl;

#if 0
    // print result
    aabb_box_t* temp_aabbs;
    hostMalloc(&temp_aabbs, primitive_count);
    hostMemcpy(temp_aabbs, d_aabbs + oibvh_internal_node_count, primitive_count);
    glm::uvec3* temp_faces;
    hostMalloc(&temp_faces, primitive_count);
    hostMemcpy(temp_faces, d_faces, primitive_count);
    for (int i = 0; i < 100; i++)
    {
        std::cout << temp_aabbs[i].minimum << "," << temp_aabbs[i].maximum << std::endl;
        std::cout << m_positions[temp_faces[i].x] << "," << m_positions[temp_faces[i].y] << "," << m_positions[temp_faces[i].z]
                  << std::endl;
    }
#endif

    const unsigned int primitiveCountNextPower2 = next_power_of_two(primitive_count);
    const unsigned int tHeight = ilog2(primitiveCountNextPower2) + 1;
    const unsigned int tLeafLev = tHeight - 1;
    unsigned int entryLevel = tLeafLev - 1;
    const unsigned int virtualLeafCount = primitiveCountNextPower2 - primitive_count;
    unsigned int entryLevelSize = oibvh_level_real_node_count(entryLevel, tLeafLev, virtualLeafCount);

    std::vector<s_param_t> scheduleParams;
    oibvh_scheduling_parameters(entryLevel, entryLevelSize, THREADS_PER_BLOCK, scheduleParams);

#if 0
    // print result
    std::cout << "scheduleParams: " << std::endl;
    for (auto param : scheduleParams)
    {
        std::cout << param.m_entryLevel << "," << param.m_realCount << "," << param.m_threadsPerGroup << ","
                  << param.m_threads << std::endl;
    }
#endif

    std::cout << "kerenl count: " << scheduleParams.size() << std::endl;

    for (int k = 0; k < scheduleParams.size(); k++)
    {
        std::cout << "kernel" << k << std::endl;
        std::cout << "  entry level: " << scheduleParams[k].m_entryLevel << std::endl;
        std::cout << "  real nodes: " << scheduleParams[k].m_realCount << std::endl;
        std::cout << "  total threads: " << scheduleParams[k].m_threads << std::endl;
        std::cout << "  group size: " << scheduleParams[k].m_threadsPerGroup << std::endl;
        std::cout << "  group count: " << scheduleParams[k].m_threads / scheduleParams[k].m_threadsPerGroup
                  << std::endl;
        elapsed_ms = kernelLaunch([&]() {
            dim3 blockSize = dim3(scheduleParams[k].m_threadsPerGroup);
            dim3 gridSize = dim3(scheduleParams[k].m_threads / scheduleParams[k].m_threadsPerGroup);
            oibvh_tree_construction_kernel<<<gridSize, blockSize>>>(scheduleParams[k].m_entryLevel,
                                                                    scheduleParams[k].m_realCount,
                                                                    primitive_count,
                                                                    scheduleParams[k].m_threadsPerGroup,
                                                                    d_aabbs);
        });
        std::cout << "  oibvh contruct kernel took: " << elapsed_ms << "ms" << std::endl;
    }

#if 0
    // print result
    aabb_box_t* temp_aabbs;
    hostMalloc(&temp_aabbs, 100);
    hostMemcpy(temp_aabbs, d_aabbs, 100);
    for (int i = 0; i < 100; i++)
    {
        std::cout << temp_aabbs[i].minimum << "," << temp_aabbs[i].maximum << std::endl;
    }
    std::cout << m_mesh->m_aabb.minimum << "," << m_mesh->m_aabb.maximum << std::endl;
#endif

    hipFree(d_positions);
    hipFree(d_faces);
    hipFree(d_aabbs);
    hipFree(d_mortons);
}